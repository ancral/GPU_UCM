#include "hip/hip_runtime.h"
//Ángel Cruz Alonso
//Ignacio Nicolás López

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
/* Time */
#include <sys/time.h>
#include <sys/resource.h>
#include "kernel.h"
#define DIMBLOCK 16
#define PI 3.141593

__global__ void NRAux(float *im, float *NR,  int height, int width) {

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int i = by*DIMBLOCK + ty;
  int j = bx*DIMBLOCK + tx;

  NR[i*width+j] = 0;

  if(i > 2 && j > 2 && i < width - 2 && j < height - 2){
    NR[i*width+j] =
				 (2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
				+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
				+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
				+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
				+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
				/159.0;

  }
}


__global__ void GradientAux(float *NR, float *Gx, float *Gy, float *G, float *phi, int height, int width){

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;


  int i = by*DIMBLOCK + ty;
  int j = bx*DIMBLOCK + tx;

  G[i*width+j] = 0;
  phi[i*width+j] = 0;

  if(i > 2 && j > 2 && i < width - 2 && j < height - 2){

    Gx[i*width+j] = 
				 (1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
				+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
				+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
				+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


    Gy[i*width+j] = 
				 ((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
				+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
				+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);

    G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
    phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

    if(fabs(phi[i*width+j])<=PI/8 )
      phi[i*width+j] = 0;
    else if (fabs(phi[i*width+j])<= 3*(PI/8))
      phi[i*width+j] = 45;
    else if (fabs(phi[i*width+j]) <= 5*(PI/8))
      phi[i*width+j] = 90;
    else if (fabs(phi[i*width+j]) <= 7*(PI/8))
      phi[i*width+j] = 135;
    else phi[i*width+j] = 0;
  }
}


__global__ void PedgeAux(float *G, float *pedge, float *phi, int height, int width){

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

 
  int i = by*DIMBLOCK + ty;
  int j = bx*DIMBLOCK + tx;

  pedge[i*width+j] = 0;

  if(i > 3 && j > 3 && i < width - 3 && j < height - 3){
    			if(phi[i*width+j] == 0){
				if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 45) {
				if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 90) {
				if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 135) {
				if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
					pedge[i*width+j] = 1;
			}
  }
} 

__global__ void thresholding(float level,float *G, float *image_out, int width, int height, float *pedge){
	float lowthres = level/2;
	float hithres  = 2*(level);
 	int bx = blockIdx.x;
  	int by = blockIdx.y;
	int ii,jj;
  	int tx = threadIdx.x;
  	int ty = threadIdx.y;


  int i = by*DIMBLOCK + ty;
  int j = bx*DIMBLOCK + tx;
	
	image_out[i*width+j] = 0;	

	if(i > 3 && j > 3 && i < width - 3 && j < height - 3){
		if(G[i*width+j]>hithres && pedge[i*width+j])
			image_out[i*width+j] = 255;
		else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
			// check neighbours 3x3
			for (ii=-1;ii<=1; ii++)
				for (jj=-1;jj<=1; jj++)
					if (G[(i+ii)*width+j+jj]>hithres)
						image_out[i*width+j] = 255;
	}

}





double get_time(){
	static struct timeval 	tv0;
	double time_, mytime;

	gettimeofday(&tv0,(struct timezone*)0);
	time_=(double)((tv0.tv_usec + (tv0.tv_sec)*1000000));
	mytime = time_/1000000;
	return(mytime);
}



void cannyGPU(float *im, float *image_out,
	float level,
	int height, int width)
{
  float *imOrig, *imFin, *Gx, *Gy, *G, *phi, *NR, *pedge;  
  double t0, t1;
  dim3 numThreads = dim3(DIMBLOCK,DIMBLOCK);
  dim3 dimBlock = dim3(ceil(height/DIMBLOCK),ceil(width/DIMBLOCK));

  
  hipMalloc((void**)&imOrig, height*width*sizeof(float));
  hipMalloc((void**)&imFin, height*width*sizeof(float));
  hipMalloc((void**)&Gx, height*width*sizeof(float));
  hipMalloc((void**)&Gy, height*width*sizeof(float));
  hipMalloc((void**)&G, height*width*sizeof(float));
  hipMalloc((void**)&phi, height*width*sizeof(float));
  hipMalloc((void**)&NR, height*width*sizeof(float));
  hipMalloc((void**)&pedge, height*width*sizeof(float));
  
  hipMemcpy(imOrig,im,height*width*sizeof(float),hipMemcpyHostToDevice);

   t0 = get_time();

  NRAux<<<dimBlock,numThreads>>>(imOrig, NR, height, width);
  GradientAux<<<dimBlock,numThreads>>>(NR, Gx, Gy, G, phi, height, width);
  PedgeAux<<<dimBlock,numThreads>>>(G, pedge, phi, height, width);
  thresholding<<<dimBlock,numThreads>>>(level,G,imFin,width,height, pedge);
  
  t1 = get_time();
  printf("GPU REAL Exection time %f ms.\n", t1-t0);
  hipMemcpy(image_out,imFin,height*width*sizeof(float),hipMemcpyDeviceToHost);
}

